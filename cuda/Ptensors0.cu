#include "hip/hip_runtime.h"
/*
This file is part of ptens, a C++/CUDA library for permutation 
equivariant message passing. 
 
Copyright (c) 2023, Imre Risi Kondor

This source code file is subject to the terms of the noncommercial 
license distributed with cnine in the file LICENSE.TXT. Commercial 
use is prohibited. All redistributed versions of this file (in 
original or modified form) must retain this copyright notice and 
must be accompanied by a verbatim copy of the license. 
*/

#ifndef _Ptensors0_cu
#define _Ptensors0_cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Ptens_base.hpp"
#include "Ltensor.hpp"
#include "AindexPackB.hpp"
#include "BatchedAindexPackB.hpp"


typedef cnine::Ltensor<float> TENSOR;
typedef cnine::Ltensor<int> ITENSOR;


__global__ void Ptensors0_reduce0_kernel(float* rarr, const int rs, const float* xarr, const int xs, 
  const int* map, const int maps, const int n){
  const int b=blockIdx.x;
  const int c=threadIdx.x;
  rarr[map[b*maps]*rs+c]+=xarr[map[b*maps+2]*xs+c];
}


__global__ void Ptensors0_broadcast0_kernel(float* rarr, const int rs, const float* xarr, const int xs, 
  const int* map, const int maps, const int* bmap){
  const int b=blockIdx.x;
  const int c=threadIdx.x;
  const int boffs=bmap[b+1];
  const int N=bmap[b+2]-bmap[b+1]-1;
  if(N==0) return;

  int target=map[bmap[boffs+1]*maps+2];
  float t=0;
  for(int s=0; s<N; s++){
    const int row=bmap[boffs+s+1];
    //if(s==0) target=map[row*maps+2];
    t+=xarr[map[row*maps]*xs+c];
  }
  rarr[target*rs+c]+=t;
}


// -----------------------------------------------------------------------------------------------------------


namespace ptens{


  void Ptensors0_reduce0_cu(const TENSOR& r, const TENSOR& x, const AindexPackB& _map, int offs, int n, const hipStream_t& stream){
    int dev=r.get_dev();
    auto& map=_map.on_device(dev);
    PTENS_ASSRT(r.get_dev()==1);
    PTENS_ASSRT(x.get_dev()==1);
    PTENS_ASSRT(r.stride(1)==1);
    PTENS_ASSRT(x.stride(1)==1);
    if(map.dim(0)==0) return;
    PTENS_CHANNEL_LIMIT(n);

    Ptensors0_reduce0_kernel<<<map.dim(0),n,0,stream>>>(r.get_arr(),r.stride(0),x.get_arr()+offs,x.stride(0),map.get_arr(),map.stride(0),n);
  }


  void Ptensors0_broadcast0_cu(const TENSOR& r, const TENSOR& x, const AindexPackB& _map, const int offs, const hipStream_t& stream){
    int dev=r.dev;
    auto& map=_map.on_device(dev);
    PTENS_ASSRT(x.dev==dev);
    PTENS_ASSRT(r.stride(1)==1);
    PTENS_ASSRT(x.stride(1)==1);

    int n=x.dim(1);
    PTENS_CHANNEL_LIMIT(n);
    int nthrd=n; //cnine::roundup(std::max(n,map.dim(1)),32);

    if(_map.n_gather_lists==0) return;
    Ptensors0_broadcast0_kernel<<<_map.n_gather_lists,nthrd,map.dim(1)*4,stream>>> 
      (r.get_arr()+offs,r.stride(0),x.get_arr(),x.stride(0),map.get_arr(),map.stride(0),
	_map.gmap_on_device(dev).get_arr());
  }


  // ---- Batched -------------------------------------------------------------------------------------------


  void Ptensors0_reduce0_cu(const TENSOR& r, const TENSOR& x, const BatchedAindexPackB& _map, int offs, int n, const hipStream_t& stream){
    int dev=r.get_dev();
    auto& map=_map.on_device(dev);
    PTENS_ASSRT(r.get_dev()==1);
    PTENS_ASSRT(x.get_dev()==1);
    PTENS_ASSRT(r.stride(1)==1);
    PTENS_ASSRT(x.stride(1)==1);
    if(map.dim(0)==0) return;
    PTENS_CHANNEL_LIMIT(n);

    Ptensors0_reduce0_kernel<<<map.dim(0),n,0,stream>>>(r.get_arr(),r.stride(0),x.get_arr()+offs,x.stride(0),map.get_arr(),map.stride(0),n);
  }

  void Ptensors0_broadcast0_cu(const TENSOR& r, const TENSOR& x, const BatchedAindexPackB& _map, const int offs, const hipStream_t& stream){
    int dev=r.dev;
    auto& map=_map.on_device(dev);
    PTENS_ASSRT(x.dev==dev);
    PTENS_ASSRT(r.stride(1)==1);
    PTENS_ASSRT(x.stride(1)==1);

    int n=x.dim(1);
    PTENS_CHANNEL_LIMIT(n);
    int nthrd=n; //cnine::roundup(std::max(n,map.dim(1)),32);

    if(_map.n_gather_lists==0) return;
    Ptensors0_broadcast0_kernel<<<_map.n_gather_lists,nthrd,map.dim(1)*4,stream>>> 
      (r.get_arr()+offs,r.stride(0),x.get_arr(),x.stride(0),map.get_arr(),map.stride(0),
	_map.gmap_on_device(dev).get_arr());
  }


  //template  void Ptensors0_reduce0_cu(const TENSOR& R, const TENSOR& x, const AindexPackB& map, int offs, int n, const hipStream_t& stream);
  //template  void Ptensors0_broadcast0_cu(const TENSOR& r, const TENSOR& x, const AindexPackB& map, const int offs, const hipStream_t& stream);

  //template  void Ptensors0_reduce0_cu(const TENSOR& R, const TENSOR& x, const BatchedAindexPackB& map, int offs, int n, const hipStream_t& stream);
  //template  void Ptensors0_broadcast0_cu(const TENSOR& r, const TENSOR& x, const BatchedAindexPackB& map, const int offs, const hipStream_t& stream);


}

#endif 
/*
__global__ void Ptensors0_reduce0_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir){
  const int i=blockIdx.x;
  const int c=threadIdx.x;
  rarr[rdir[2*i]+c]+=xarr[xdir[2*i]+c];
}
*/
/*
__global__ void Ptensors0_broadcast0_kernel(float* xarr, const int* xdir, const float* rarr, const int* rdir){
  const int i=blockIdx.x;
  const int c=threadIdx.x;
  xarr[xdir[2*i]+c]+=rarr[rdir[2*i]+c];
}
*/
/*
__global__ void Ptensors0_gather_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir, const float* marr, const int* mdir){
  const int i=blockIdx.x;
  const int c=threadIdx.x;

  const int moffs=mdir[2*i];
  const int N=mdir[2*i+1]/2;
  float t=0;
  for(int j=0; j<N; j++){
    const int jix=*reinterpret_cast<const int*>(marr+moffs+2*j);
    const int jweight=marr[moffs+2*j+1];
    t+=jweight*xarr[xdir[2*jix]+c];
  }
  rarr[rdir[2*i]+c]+=t;
}
*/
  /*
  void Ptensors0_reduce0_cu(Ptensors0& R, const Ptensors0& x, int offs, int n, const hipStream_t& stream){
    int dev=R.get_dev();
    PTENS_ASSRT(R.get_dev()==1);
    PTENS_ASSRT(x.get_dev()==1);
    if(R.size()==0) return;
    Ptensors0_reduce0_kernel<<<R.size(),n,0,stream>>>(R.arrg,R.dir.garr(dev),x.arrg+offs,x.dir.garr(dev));
  }
  */
  /*
  void Ptensors0_broadcast0_cu(Ptensors0& x, const Ptensors0& R, const int offs, const hipStream_t& stream){
    int dev=R.get_dev();
    PTENS_ASSRT(R.get_dev()==1);
    PTENS_ASSRT(x.get_dev()==1);
    if(R.size()==0) return;
    Ptensors0_broadcast0_kernel<<<R.size(),x.nc,0,stream>>>
      (x.arrg+offs,x.dir.garr(dev),R.arrg,R.dir.garr(dev));
  }
  */
